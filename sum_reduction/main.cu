#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <fstream>

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "utils.h"

#include "reduce.h"
extern "C" {
#include <pinpool.h>
#include <filemap.h>
}

using namespace std;

void generate_input(unsigned int* input, unsigned int input_len)
{
	for (unsigned int i = 0; i < input_len; ++i)
	{
		input[i] = i;
	}
}

unsigned int cpu_simple_sum(unsigned int* h_in, unsigned int h_in_len)
{
	unsigned int total_sum = 0;

	for (unsigned int i = 0; i < h_in_len; ++i)
	{
		total_sum = total_sum + h_in[i];
	}

	return total_sum;
}

int main()
{
	// Set up clock for timing comparisons
	std::clock_t start;
	double duration;
        int check = pinpool_init(1, 67108864);

	for (int k = 1; k < 28; ++k)
	{
		unsigned int h_in_len = (1 << k);
		//unsigned int h_in_len = 2048;
		std::cout << "h_in_len: " << h_in_len << std::endl;
		unsigned int* t_in = new unsigned int[h_in_len];
		generate_input(t_in, h_in_len);
                FILE *testfile_out = fopen("/home/gpurocks/data_mysql/testfile.data", "wb");
                //ofstream testfile_out("/home/gpurocks/data_mysql/testfile.data", ios::out);
                for(unsigned int i = 0; i < h_in_len; i++)
                  fwrite((t_in + i), sizeof(unsigned int), 1, testfile_out);
                   //testfile_out << *(t_in + i);
                delete[] t_in;
                //testfile_out.close();
                fclose(testfile_out);

                start = std::clock();
                FILE *testfile_in = fopen("/home/gpurocks/data_mysql/testfile.data", "rb");
                //ifstream testfile_in("/home/gpurocks/data_mysql/testfile.data", ios::in);
                unsigned int* h_in = new unsigned int[h_in_len];
                for(unsigned int i = 0; i < h_in_len; i++)
                  if(!fread((h_in + i), sizeof(unsigned int), 1, testfile_in)) return 0;
                  //testfile_in >> *(h_in + i);
                //testfile_in.close();
                fclose(testfile_in);
 
                duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;              
                std::cout << "Read file time : " << duration << " s" << std::endl;

		// Do CPU sum for reference
		start = std::clock();
		unsigned int cpu_total_sum = cpu_simple_sum(h_in, h_in_len);
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << cpu_total_sum << std::endl;
		std::cout << "CPU time: " << duration << " s" << std::endl;

		// Do GPU scan
		start = std::clock();
                unsigned int* d_in;
                hipMalloc(&d_in, sizeof(unsigned int) * h_in_len);
                hipMemcpy(d_in, h_in, sizeof(unsigned int) * h_in_len, hipMemcpyHostToDevice);

		unsigned int gpu_total_sum = gpu_sum_reduce(d_in, h_in_len);
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << gpu_total_sum << std::endl;
		std::cout << "GPU time: " << duration << " s" << std::endl;
                hipFree(d_in);
                delete[] h_in;

                // Do GPU_DMA Scan
                start = std::clock();
                unsigned int* dma_in;     
                struct filemap* testfile_dma = filemap_open_cuda("/home/gpurocks/data_mysql/testfile.data");
                dma_in = (unsigned int*)testfile_dma->data;
                    
                unsigned int gpu_dma_total_sum = gpu_sum_reduce_dma(dma_in, h_in_len);
                duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
                std::cout << gpu_dma_total_sum << std::endl;
                std::cout << "GPU_DMA time: " << duration << " s" << std::endl;
                filemap_free(testfile_dma);


		bool match = (cpu_total_sum == gpu_total_sum) && (cpu_total_sum == gpu_dma_total_sum);
		std::cout << "Match: " << match << std::endl;

		std::cout << std::endl;
	}
}
